#include "hip/hip_runtime.h"
#include "NMR_cpmg_cuda.h"

__global__ void CPMG_walk_test(void)
{    
    
}

// function to call GPU kernel to execute
// walker's "walk" method in Graphics Processing Unit
void NMR_cpmg::image_simulation_cuda()
{
    // define kernel launch flags
    const bool applyField = (this->internalField != NULL);
    string bc = this->model.getBoundaryCondition();
    const bool bcMirrorFlag = (bc == "mirror");
    const bool interpFlag = (*this).getCpmgConfig().getInterpolateField();
    cout << "- starting RW-CPMG simulation (in GPU) [bc:" << bc;
    if(applyField and (*this).getCpmgConfig().getInterpolateField()) cout << "+3Linterp";
    cout << "]...";
    
    bool time_verbose = this->CPMG_config.getTimeVerbose();
    double reset_time = 0.0;
    double copy_time = 0.0;
    double kernel_time = 0.0;
    double buffer_time = 0.0;
    double reduce_time = 0.0;
    
    double tick = omp_get_wtime();
    if(this->model.getRwnmrConfig().getOpenMPUsage())
    {
        // set omp variables for parallel loop throughout walker list
        const int num_cpu_threads = omp_get_max_threads();
        const int loop_size = this->model.getWalkers()->size();
        int loop_start, loop_finish;

        #pragma omp parallel private(loop_start, loop_finish) 
        {
            const int thread_id = omp_get_thread_num();
            ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
            loop_start = looper.getStart();
            loop_finish = looper.getFinish(); 

            for (uint id = loop_start; id < loop_finish; id++)
            {
                (*this->model.getWalkers())[id].resetPosition();
                (*this->model.getWalkers())[id].resetSeed();
                (*this->model.getWalkers())[id].resetEnergy();
            }
        }
    } else
    {
        // reset walker's initial state 
        for (uint id = 0; id < this->model.getWalkers()->size(); id++)
        {
            (*this->model.getWalkers())[id].resetPosition();
            (*this->model.getWalkers())[id].resetSeed();
            (*this->model.getWalkers())[id].resetEnergy();
        }
    }

    // reset vector to store energy decay
    (*this).resetSignal();
    this->signalAmps.reserve(this->model.getNumberOfEchoes() + 1); // '+1' to accomodate time 0.0

    // get initial energy global state
    double energySum = ((double) this->model.getWalkers()->size()) * (*this->model.getWalkers())[0].getEnergy();
    this->signalAmps.push_back(energySum);

    reset_time += omp_get_wtime() - tick;


    // CUDA event recorder to measure computation time in device
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // integer values for sizing issues
    uint numberOfWalkers = this->model.getNumberOfWalkers();
    uint bitBlockColumns = this->model.getBitBlock()->getBlockColumns();
    uint bitBlockRows = this->model.getBitBlock()->getBlockRows();
    uint numberOfBitBlocks = this->model.getBitBlock()->getNumberOfBlocks();
    int imageColumns = this->model.getBitBlock()->getImageColumns();
    int imageRows = this->model.getBitBlock()->getImageRows();
    int imageDepth = this->model.getBitBlock()->getImageDepth();
    uint shiftConverter = log2(this->model.getVoxelDivision());

    uint numberOfEchoes = this->model.getNumberOfEchoes();
    uint stepsPerEcho = this->model.getStepsPerEcho();
    uint echoesPerKernel = this->model.getRwnmrConfig().getEchoesPerKernel();
    uint kernelCalls = (uint) ceil(numberOfEchoes / (double) echoesPerKernel);


    // Internal field parameters
    double *field = (*this).getInternalFieldData();
    long fieldSize = (*this).getInternalFieldSize();
    double timeInterval = 1.0e-3 * this->model.getTimeInterval(); 
    double gamma = 1.0e+06 * this->model.getGiromagneticRatio();
    
    // define parameters for CUDA kernel launch: blockDim, gridDim etc
    uint threadsPerBlock = this->model.getRwnmrConfig().getThreadsPerBlock();
    uint blocksPerKernel = this->model.getRwnmrConfig().getBlocks();
    uint walkersPerKernel = threadsPerBlock * blocksPerKernel;

    // treat case when only one kernel is needed
    if (numberOfWalkers < walkersPerKernel)
    {
        blocksPerKernel = (uint)ceil((double)(numberOfWalkers) / (double)(threadsPerBlock));

        // blocks per kernel should be multiple of 2
        if (blocksPerKernel % 2 == 1)
        {
            blocksPerKernel += 1;
        }

        walkersPerKernel = threadsPerBlock * blocksPerKernel;
    }

    // Walker packs == groups of walkers in the same kernel
    // all threads in a pack represent a walker in the NMR simulation
    // But, in the last pack, some threads may be idle
    uint numberOfWalkerPacks = (numberOfWalkers / walkersPerKernel) + 1;
    uint lastWalkerPackSize = numberOfWalkers % walkersPerKernel;
    uint lastWalkerPackTail = walkersPerKernel - lastWalkerPackSize;
    uint energyArraySize = walkersPerKernel;
    uint energyCollectorSize = (blocksPerKernel / 2);

    // Copy bitBlock3D data from host to device (only once)
    // assign pointer to bitBlock datastructure
    uint64_t *bitBlock;
    bitBlock = this->model.getBitBlock()->getBlocks();
    uint64_t *d_bitBlock;
    hipMalloc((void **)&d_bitBlock, numberOfBitBlocks * sizeof(uint64_t));
    hipMemcpy(d_bitBlock, bitBlock, numberOfBitBlocks * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Host and Device memory data allocation
    // pointers used in host array conversion
    int *walker_px = MemAllocator::mallocIntArray(walkersPerKernel);
    int *walker_py = MemAllocator::mallocIntArray(walkersPerKernel);
    int *walker_pz = MemAllocator::mallocIntArray(walkersPerKernel);
    double *penalty = MemAllocator::mallocDoubleArray(walkersPerKernel);
    double *pAlive = MemAllocator::mallocDoubleArray(walkersPerKernel);
    double *phase = MemAllocator::mallocDoubleArray(walkersPerKernel);
    double *energy = MemAllocator::mallocDoubleArray(echoesPerKernel * energyArraySize);
    double *energyCollector = MemAllocator::mallocDoubleArray(echoesPerKernel * energyCollectorSize);
    uint64_t *seed = MemAllocator::mallocUInt64Array(walkersPerKernel);
    
    // temporary array to collect energy contributions for each echo in a kernel
    double *temp_globalEnergy = MemAllocator::mallocDoubleArray((uint)echoesPerKernel);
    double *h_globalEnergy = MemAllocator::mallocDoubleArray(kernelCalls * echoesPerKernel);

    tick = omp_get_wtime();
    for (uint echo = 0; echo < numberOfEchoes; echo++)
    {
        h_globalEnergy[echo] = 0.0;
    }
    buffer_time += omp_get_wtime() - tick;

    // Declaration of pointers to device data arrays
    int *d_walker_px;
    int *d_walker_py;
    int *d_walker_pz;
    double *d_penalty;
    double *d_pAlive;
    double *d_phase;
    double *d_field;
    double *d_energy;
    double *d_energyCollector;
    uint64_t *d_seed;

    // Memory allocation in device for data arrays
    hipMalloc((void **)&d_walker_px, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_walker_py, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_walker_pz, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_penalty, walkersPerKernel * sizeof(double));
    if(applyField) hipMalloc((void **)&d_pAlive, walkersPerKernel * sizeof(double));
    if(applyField) hipMalloc((void **)&d_phase, walkersPerKernel * sizeof(double));
    hipMalloc((void **)&d_energy, echoesPerKernel * energyArraySize * sizeof(double));
    hipMalloc((void **)&d_energyCollector, echoesPerKernel * energyCollectorSize * sizeof(double));
    hipMalloc((void **)&d_seed, walkersPerKernel * sizeof(uint64_t));
    
    if(applyField)
    {
        hipMalloc((void **)&d_field, fieldSize * sizeof(double));
        hipMemcpy(d_field, field, fieldSize * sizeof(double), hipMemcpyHostToDevice);
    }

    tick = omp_get_wtime();
    for (uint i = 0; i < energyArraySize * echoesPerKernel; i++)
    {
        energy[i] = 0.0;
    }
    buffer_time += omp_get_wtime() - tick;

    for (uint packId = 0; packId < (numberOfWalkerPacks - 1); packId++)
    {
        // set offset in walkers vector
        uint packOffset = packId * walkersPerKernel;

        // Host data copy
        // copy original walkers' data to temporary host arrays
        tick = omp_get_wtime();
        if(this->model.getRwnmrConfig().getOpenMPUsage())
        {
            // set omp variables for parallel loop throughout walker list
            const int num_cpu_threads = omp_get_max_threads();
            const int loop_size = walkersPerKernel;
            int loop_start, loop_finish;

            #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, penalty, energy, seed) private(loop_start, loop_finish) 
            {
                const int thread_id = omp_get_thread_num();
                ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                loop_start = looper.getStart();
                loop_finish = looper.getFinish(); 

                for (uint i = loop_start; i < loop_finish; i++)
                {
                    walker_px[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionX();
                    walker_py[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionY();
                    walker_pz[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionZ();
                    penalty[i] = (*this->model.getWalkers())[i + packOffset].getDecreaseFactor();
                    pAlive[i] = 1.0;
                    phase[i] = 0.0;
                    energy[i + ((echoesPerKernel - 1) * energyArraySize)] = (*this->model.getWalkers())[i + packOffset].getEnergy();
                    seed[i] = (*this->model.getWalkers())[i + packOffset].getInitialSeed();
                }
            }
        } else
        {            
            for (uint i = 0; i < walkersPerKernel; i++)
            {
                walker_px[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionX();
                walker_py[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionY();
                walker_pz[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionZ();
                penalty[i] = (*this->model.getWalkers())[i + packOffset].getDecreaseFactor();
                pAlive[i] = 1.0;    
                phase[i] = 0.0;
                energy[i + ((echoesPerKernel - 1) * energyArraySize)] = (*this->model.getWalkers())[i + packOffset].getEnergy();
                seed[i] = (*this->model.getWalkers())[i + packOffset].getInitialSeed();
            }
        }  
        buffer_time += omp_get_wtime() - tick;      

        // Device data copy
        // copy host data to device
        tick = omp_get_wtime();
        hipMemcpy(d_walker_px, walker_px, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_walker_py, walker_py, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_walker_pz, walker_pz, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_penalty, penalty, walkersPerKernel * sizeof(double), hipMemcpyHostToDevice);
        if(applyField) hipMemcpy(d_pAlive, pAlive, walkersPerKernel * sizeof(double), hipMemcpyHostToDevice);
        if(applyField) hipMemcpy(d_phase, phase, walkersPerKernel * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_energy, energy, echoesPerKernel * energyArraySize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_seed, seed, walkersPerKernel * sizeof(uint64_t), hipMemcpyHostToDevice);
        copy_time += omp_get_wtime() - tick;

        // Launch kernel for GPU computation
        for (uint kernelId = 0; kernelId < kernelCalls; kernelId++)
        {
            // define echo offset
            uint echoOffset = kernelId * echoesPerKernel;
            uint echoes = echoesPerKernel;

            /* 
                Call adequate RW kernel depending on the chosen boundary treatment
            */
            tick = omp_get_wtime();
            if(applyField)
            {
                if(bcMirrorFlag and interpFlag)
                {
                    CPMG_walk_field_interp<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        walkersPerKernel, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else if(bcMirrorFlag and !interpFlag)
                {
                    CPMG_walk_field<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        walkersPerKernel, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else if(!bcMirrorFlag and interpFlag)
                {
                    CPMG_walk_field_interp<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        walkersPerKernel, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else 
                {
                    CPMG_walk_field<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        walkersPerKernel, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
            } 
            else
            {
                if(bcMirrorFlag)
                {
                    CPMG_walk<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px,
                        d_walker_py,
                        d_walker_pz,
                        d_penalty,
                        d_energy,
                        d_seed,
                        d_bitBlock,
                        bitBlockColumns,
                        bitBlockRows,
                        walkersPerKernel,
                        energyArraySize,
                        echoes,
                        stepsPerEcho,
                        imageColumns,
                        imageRows,
                        imageDepth,
                        shiftConverter
                    );
                }
                else
                {
                    CPMG_walk<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px,
                        d_walker_py,
                        d_walker_pz,
                        d_penalty,
                        d_energy,
                        d_seed,
                        d_bitBlock,
                        bitBlockColumns,
                        bitBlockRows,
                        walkersPerKernel,
                        energyArraySize,
                        echoes,
                        stepsPerEcho,
                        imageColumns,
                        imageRows,
                        imageDepth,
                        shiftConverter
                    );
                }  
            }

            hipDeviceSynchronize();
            kernel_time += omp_get_wtime() - tick;

            
            // launch globalEnergy "reduce" kernel
            tick = omp_get_wtime();
            CPMG_energyReduce<<<blocksPerKernel / 2,
                                threadsPerBlock,
                                threadsPerBlock * sizeof(double)>>>(d_energy,
                                                                    d_energyCollector,
                                                                    energyArraySize,
                                                                    energyCollectorSize,
                                                                    echoesPerKernel);
            hipDeviceSynchronize();
            reduce_time += omp_get_wtime() - tick;

            // copy data from gatherer array
            tick = omp_get_wtime();
            hipMemcpy(energyCollector,
                       d_energyCollector,
                       echoesPerKernel * energyCollectorSize * sizeof(double),
                       hipMemcpyDeviceToHost);
            copy_time += omp_get_wtime() - tick;

            //last reduce is done in CPU parallel-style using openMP
            tick = omp_get_wtime();
            CPMG_reduce_omp(temp_globalEnergy, energyCollector, echoesPerKernel, blocksPerKernel / 2);
            reduce_time += omp_get_wtime() - tick;

            // copy data from temporary array to NMR_Simulation2D "globalEnergy" vector class member
            tick = omp_get_wtime();
            for (uint echo = 0; echo < echoesPerKernel; echo++)
            {
                h_globalEnergy[echo + echoOffset] += temp_globalEnergy[echo];
            }
            buffer_time += omp_get_wtime() - tick;

            // recover last positions
            tick = omp_get_wtime();
            hipMemcpy(walker_px, d_walker_px, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_py, d_walker_py, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_pz, d_walker_pz, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);      
            copy_time = omp_get_wtime() - tick;

            tick = omp_get_wtime();
            if(this->model.getRwnmrConfig().getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = walkersPerKernel;
                int loop_start, loop_finish;

                #pragma omp parallel shared(walker_px, walker_py, walker_pz, packOffset) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]);
                    }
                }
            } else
            {
                for (uint i = 0; i < walkersPerKernel; i++)
                {
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionX(walker_px[i]);
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionY(walker_py[i]);
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionZ(walker_pz[i]); 
                }
            }
            buffer_time += omp_get_wtime() - tick;  
        }
    }

    if (lastWalkerPackSize > 0)
    {
        // last Walker pack is done explicitly
        // set offset in walkers vector
        uint packOffset = (numberOfWalkerPacks - 1) * walkersPerKernel;

        // Host data copy
        // copy original walkers' data to temporary host arrays
        tick = omp_get_wtime();
        if(this->model.getRwnmrConfig().getOpenMPUsage())
        {
            // set omp variables for parallel loop throughout walker list
            const int num_cpu_threads = omp_get_max_threads();
            const int loop_size = lastWalkerPackSize;
            int loop_start, loop_finish;

            #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, penalty, energy, seed) private(loop_start, loop_finish) 
            {
                const int thread_id = omp_get_thread_num();
                ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                loop_start = looper.getStart();
                loop_finish = looper.getFinish(); 

                for (uint i = loop_start; i < loop_finish; i++)
                {
                    walker_px[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionX();
                    walker_py[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionY();
                    walker_pz[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionZ();
                    penalty[i] = (*this->model.getWalkers())[i + packOffset].getDecreaseFactor();
                    pAlive[i] = 1.0;
                    phase[i] = 0.0;
                    energy[i + ((echoesPerKernel - 1) * energyArraySize)] = (*this->model.getWalkers())[i + packOffset].getEnergy();
                    seed[i] = (*this->model.getWalkers())[i + packOffset].getInitialSeed();
                }
            }
        } else
        {            
            for (uint i = 0; i < lastWalkerPackSize; i++)
            {
                walker_px[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionX();
                walker_py[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionY();
                walker_pz[i] = (*this->model.getWalkers())[i + packOffset].getInitialPositionZ();
                penalty[i] = (*this->model.getWalkers())[i + packOffset].getDecreaseFactor();
                pAlive[i] = 1.0;
                phase[i] = 0.0;
                energy[i + ((echoesPerKernel - 1) * energyArraySize)] = (*this->model.getWalkers())[i + packOffset].getEnergy();
                seed[i] = (*this->model.getWalkers())[i + packOffset].getInitialSeed();
            }
        }   

        // complete energy array data
        for (uint echo = 0; echo < echoesPerKernel; echo++)
        {
            for (uint i = 0; i < lastWalkerPackTail; i++)
            {
                {
                    energy[i + lastWalkerPackSize + (echo * energyArraySize)] = 0.0;
                }
            }
        }
        buffer_time += omp_get_wtime() - tick;
        

        // Device data copy
        // copy host data to device
        tick = omp_get_wtime();
        hipMemcpy(d_walker_px, walker_px, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_walker_py, walker_py, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_walker_pz, walker_pz, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_penalty, penalty, lastWalkerPackSize * sizeof(double), hipMemcpyHostToDevice);
        if(applyField) hipMemcpy(d_pAlive, pAlive, lastWalkerPackSize * sizeof(double), hipMemcpyHostToDevice);
        if(applyField) hipMemcpy(d_phase, phase, lastWalkerPackSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_energy, energy, echoesPerKernel * energyArraySize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_seed, seed, lastWalkerPackSize * sizeof(uint64_t), hipMemcpyHostToDevice);
        copy_time += omp_get_wtime() - tick;
        

        // Launch kernel for GPU computation
        for (uint kernelId = 0; kernelId < kernelCalls; kernelId++)
        {
            // define echo offset
            uint echoOffset = kernelId * echoesPerKernel;
            uint echoes = echoesPerKernel;

            /* 
                Call adequate RW kernel depending on the chosen boundary treatment
            */
            tick = omp_get_wtime();
            if(applyField)
            {
                if(bcMirrorFlag and interpFlag)
                {
                    CPMG_walk_field_interp<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        lastWalkerPackSize, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else if(bcMirrorFlag and !interpFlag)
                {
                    CPMG_walk_field<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        lastWalkerPackSize, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else if(!bcMirrorFlag and interpFlag)
                {
                    CPMG_walk_field_interp<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        lastWalkerPackSize, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
                else 
                {
                    CPMG_walk_field<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px, 
                        d_walker_py, 
                        d_walker_pz, 
                        d_penalty, 
                        d_energy,
                        d_pAlive,
                        d_phase, 
                        d_seed, 
                        d_bitBlock, 
                        bitBlockColumns, 
                        bitBlockRows, 
                        lastWalkerPackSize, 
                        energyArraySize, 
                        echoes, 
                        stepsPerEcho, 
                        imageColumns, 
                        imageRows, 
                        imageDepth, 
                        shiftConverter,
                        gamma,
                        timeInterval,
                        d_field
                    );
                }
            } 
            else
            {
                if(bcMirrorFlag)
                {
                    CPMG_walk<true><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px,
                        d_walker_py,
                        d_walker_pz,
                        d_penalty,
                        d_energy,
                        d_seed,
                        d_bitBlock,
                        bitBlockColumns,
                        bitBlockRows,
                        lastWalkerPackSize,
                        energyArraySize,
                        echoes,
                        stepsPerEcho,
                        imageColumns,
                        imageRows,
                        imageDepth,
                        shiftConverter
                    );
                }
                else
                {
                    CPMG_walk<false><<<blocksPerKernel, threadsPerBlock>>>(
                        d_walker_px,
                        d_walker_py,
                        d_walker_pz,
                        d_penalty,
                        d_energy,
                        d_seed,
                        d_bitBlock,
                        bitBlockColumns,
                        bitBlockRows,
                        lastWalkerPackSize,
                        energyArraySize,
                        echoes,
                        stepsPerEcho,
                        imageColumns,
                        imageRows,
                        imageDepth,
                        shiftConverter
                    );
                }
            }
        
            hipDeviceSynchronize();
            kernel_time += omp_get_wtime() - tick;

            // launch globalEnergy "reduce" kernel
            tick = omp_get_wtime();
            CPMG_energyReduce<<<blocksPerKernel / 2,
                                threadsPerBlock,
                                threadsPerBlock * sizeof(double)>>>(d_energy,
                                                                    d_energyCollector,
                                                                    energyArraySize,
                                                                    energyCollectorSize,
                                                                    echoesPerKernel);
            hipDeviceSynchronize();
            reduce_time += omp_get_wtime() - tick;

            // copy data from gatherer array
            tick = omp_get_wtime();
            hipMemcpy(energyCollector,
                       d_energyCollector,
                       echoesPerKernel * energyCollectorSize * sizeof(double),
                       hipMemcpyDeviceToHost);
            copy_time += omp_get_wtime() - tick;

            //last reduce is done in CPU parallel-style using openMP
            tick = omp_get_wtime();
            CPMG_reduce_omp(temp_globalEnergy, energyCollector, echoesPerKernel, blocksPerKernel / 2);
            reduce_time += omp_get_wtime() - tick;

            // copy data from temporary array
            tick = omp_get_wtime();
            for (uint echo = 0; echo < echoesPerKernel; echo++)
            {
                h_globalEnergy[echo + echoOffset] += temp_globalEnergy[echo];
            }
            buffer_time += omp_get_wtime() - tick;

            // recover last positions
            tick = omp_get_wtime();
            hipMemcpy(walker_px, d_walker_px, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_py, d_walker_py, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_pz, d_walker_pz, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);      
            copy_time += omp_get_wtime() - tick;

            tick = omp_get_wtime();
            if(this->model.getRwnmrConfig().getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = lastWalkerPackSize;
                int loop_start, loop_finish;

                #pragma omp parallel shared(walker_px, walker_py, walker_pz, packOffset) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                        (*this->model.getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]);
                    }
                }
            } else
            {
                for (uint i = 0; i < lastWalkerPackSize; i++)
                {
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionX(walker_px[i]);
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionY(walker_py[i]);
                    (*this->model.getWalkers())[i + packOffset].setCurrentPositionZ(walker_pz[i]);            
                }
            }
            buffer_time += omp_get_wtime() - tick;  
        }
    }

    // insert to object energy values computed in gpu
    tick = omp_get_wtime();
    for (uint echo = 0; echo < numberOfEchoes; echo++)
    {
        this->signalAmps.push_back(h_globalEnergy[echo]);
    }
    buffer_time += omp_get_wtime() - tick;

    // free pointers in host
    free(walker_px);
    free(walker_py);
    free(walker_pz);
    free(penalty);
    free(pAlive);
    free(phase);
    free(h_globalEnergy);
    free(energy);
    free(energyCollector);
    free(temp_globalEnergy);
    free(seed);

    // and direct them to NULL
    walker_px = NULL;
    walker_py = NULL;
    walker_pz = NULL;
    penalty = NULL;
    pAlive = NULL;
    phase = NULL;
    h_globalEnergy = NULL;
    energy = NULL;
    energyCollector = NULL;
    temp_globalEnergy = NULL;
    seed = NULL;

    // also direct the bitBlock pointer created in this context
    // (original data is kept safe)
    bitBlock = NULL;
    field = NULL;

    // free device global memory
    hipFree(d_walker_px);
    hipFree(d_walker_py);
    hipFree(d_walker_pz);
    hipFree(d_penalty);
    if(applyField) hipFree(d_pAlive);
    if(applyField) hipFree(d_phase);
    if(applyField) hipFree(d_field);
    hipFree(d_energy);
    hipFree(d_energyCollector);
    hipFree(d_seed);
    hipFree(d_bitBlock);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Done.\nCpu/Gpu elapsed time: " << elapsedTime * 1.0e-3 << " s" << endl;
    hipDeviceReset();

    if(time_verbose)
    {
        cout << "--- Time analysis ---" << endl;
        cout << "cpu data reset:    \t" << reset_time << " s" << endl;
        cout << "cpu data buffer:   \t" << buffer_time << " s" << endl;
        cout << "gpu data copy:     \t" << copy_time << " s" << endl;
        cout << "gpu kernel launch: \t" << kernel_time << " s" << endl;
        cout << "gpu reduce launch: \t" << reduce_time << " s" << endl;
        cout << "---------------------" << endl;
    }
}


/////////////////////////////////////////////////////////////////////
//////////////////////// HOST FUNCTIONS ///////////////////////////
/////////////////////////////////////////////////////////////////////
void CPMG_reduce_omp(double *temp_collector, double *array, int numberOfEchoes, uint arraySizePerEcho)
{
    // declaring shared variables
    uint offset;
    double arraySum;
    uint id;

    for (int echo = 0; echo < numberOfEchoes; echo++)
    {
        arraySum = 0.0;
        offset = (echo * arraySizePerEcho);

// #pragma omp parallel for reduction(+ \
//                                    : arraySum) private(id) shared(array, offset)
        for (id = 0; id < arraySizePerEcho; id++)
        {
            arraySum += array[id + offset];
        }

        temp_collector[echo] = arraySum;
    }
}

/////////////////////////////////////////////////////////////////////
//////////////////////// DEVICE FUNCTIONS ///////////////////////////
/////////////////////////////////////////////////////////////////////

__device__ direction computeNextDirection_CPMG(uint64_t &seed)
{
    // generate random number using xorshift algorithm
    xorshift64_state xor_state;
    xor_state.a = seed;
    seed = xorShift64_CPMG(&xor_state);
    uint64_t rand = seed;

    // set direction based on the random number
    direction nextDirection = (direction)(mod6_CPMG(rand) + 1);
    return nextDirection;
}

__device__ uint64_t xorShift64_CPMG(struct xorshift64_state *state)
{
    uint64_t x = state->a;
    x ^= x << 13;
    x ^= x >> 7;
    x ^= x << 17;
    return state->a = x;
}

__device__ uint64_t mod6_CPMG(uint64_t a)
{
    while (a > 11)
    {
        int s = 0; /* accumulator for the sum of the digits */
        while (a != 0)
        {
            s = s + (a & 7);
            a = (a >> 2) & -2;
        }
        a = s;
    }
    /* note, at this point: a < 12 */
    if (a > 5)
        a = a - 6;
    return a;
}

__device__ void computeNextPosition_CPMG(int &walker_px,
                                       int &walker_py,
                                       int &walker_pz,
                                       direction nextDirection,
                                       int &next_x,
                                       int &next_y,
                                       int &next_z)
{
    next_x = walker_px;
    next_y = walker_py;
    next_z = walker_pz;

    switch (nextDirection)
    {
    case North:
        next_y = next_y - 1;
        break;

    case West:
        next_x = next_x - 1;
        break;

    case South:
        next_y = next_y + 1;
        break;

    case East:
        next_x = next_x + 1;
        break;

    case Up:
        next_z = next_z + 1;
        break;

    case Down:
        next_z = next_z - 1;
        break;
    }
}

__device__ bool checkNextPosition_CPMG(int next_x,
                                     int next_y,
                                     int next_z,
                                     const uint64_t *bitBlock,
                                     const int bitBlockColumns,
                                     const int bitBlockRows)
{
    int blockIndex = findBlockIndex_CPMG(next_x, next_y, next_z, bitBlockColumns, bitBlockRows);
    int nextBit = findBitIndex_CPMG(next_x, next_y, next_z);
    uint64_t nextBlock = bitBlock[blockIndex];

    return (!checkIfBlockBitIsWall_CPMG(nextBlock, nextBit));
};

__device__ int findBlockIndex_CPMG(int next_x, int next_y, int next_z, int bitBlockColumns, int bitBlockRows)
{
    // "x >> 2" is like "x / 4" in bitwise operation
    int block_x = next_x >> 2;
    int block_y = next_y >> 2;
    int block_z = next_z >> 2;
    int blockIndex = block_x + block_y * bitBlockColumns + block_z * (bitBlockColumns * bitBlockRows);

    return blockIndex;
}

__device__ int findBitIndex_CPMG(int next_x, int next_y, int next_z)
{
    // "x & (n - 1)" is lise "x % n" in bitwise operation
    int bit_x = next_x & (COLUMNSPERBLOCK3D - 1);
    int bit_y = next_y & (ROWSPERBLOCK3D - 1);
    int bit_z = next_z & (DEPTHPERBLOCK3D - 1);
    // "x << 3" is like "x * 8" in bitwise operation
    int bitIndex = bit_x + (bit_y << 2) + ((bit_z << 2) << 2);

    return bitIndex;
}

__device__ bool checkIfBlockBitIsWall_CPMG(uint64_t nextBlock, int nextBit)
{
    return ((nextBlock >> nextBit) & 1ull);
}

__device__ int convertLocalToGlobal_CPMG(int _localPos, uint _shiftConverter)
{
    return (_localPos >> _shiftConverter);
}

__device__ int bcMapPeriodic_CPMG_old(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos;

    globalPos = convertLocalToGlobal_CPMG(_localPos, _shiftConverter);
    globalPos = globalPos % _dimSize;
    if(globalPos < 0) return globalPos + _dimSize;
    else return globalPos;
}

__device__ int bcMapPeriodic_CPMG(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos;
    globalPos = ( convertLocalToGlobal_CPMG(_localPos, _shiftConverter) ) % _dimSize;
    globalPos += (((int) (globalPos < 0)) * _dimSize);
    return globalPos;
}

__device__ int bcMapMirror_CPMG_old(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos;
    int imgPos;
    int mirror;
    int antimirror;

    globalPos = convertLocalToGlobal_CPMG(_localPos, _shiftConverter);
    imgPos = globalPos % _dimSize;    
    if(imgPos < 0) imgPos += _dimSize;
    if(globalPos > 0) mirror = (globalPos / _dimSize) % 2;
    else mirror = ((-globalPos - 1 + _dimSize) / _dimSize) % 2; 
    antimirror = (mirror + 1) % 2;
    return (antimirror * imgPos) + (mirror * (_dimSize - 1 - imgPos));
}

__device__ int bcMapMirror_CPMG(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos = convertLocalToGlobal_CPMG(_localPos, _shiftConverter);
    int imgPos = (globalPos % _dimSize); 
    imgPos += (((int) (imgPos < 0)) * _dimSize);
    int mirror = ((isPositive_cpmg(globalPos)*globalPos) + (((int) !(globalPos > 0)) * (-globalPos-1+_dimSize)))/_dimSize; 
    mirror = (mirror & 1);
    return (((mirror + 1) & 1) * imgPos) + (mirror * (_dimSize - 1 - imgPos));
}

__device__ int isPositive_cpmg(int x) 
{
   return !((x&(1<<31)) | !x);
}

__device__ int isZero_cpmg(int x) 
{
   return !x;
}

__device__ long getFieldIndex(int _x, int _y, int _z, int _rowScale, int _depthScale)
{ 
    return (_x + (_y * _rowScale) + (_z * _depthScale)); 
}

__device__ void getNeighbors_periodic(Neighborhood &n, int x, int y, int z, int stride, uint shiftConverter, int cols, int rows, int depth)
{
    int x0 = (x >> shiftConverter) - (int) ( ((x & (stride-1)) < (stride>>1)) ); // se x for menor do que x0 tem que diminuir 1
    int x1 = (x >> shiftConverter) + (int) (!((x & (stride-1)) < (stride>>1)) ); // se x for maior do que x0 tem que aumentar 1
    int y0 = (y >> shiftConverter) - (int) ( ((y & (stride-1)) < (stride>>1)) ); 
    int y1 = (y >> shiftConverter) + (int) (!((y & (stride-1)) < (stride>>1)) );
    int z0 = (z >> shiftConverter) - (int) ( ((z & (stride-1)) < (stride>>1)) ); 
    int z1 = (z >> shiftConverter) + (int) (!((z & (stride-1)) < (stride>>1)) );

    n.p000.x = bcMapPeriodic_CPMG(x0,0,cols); 
    n.p000.y = bcMapPeriodic_CPMG(y0,0,rows); 
    n.p000.z = bcMapPeriodic_CPMG(z0,0,depth); 
    
    n.p100.x = bcMapPeriodic_CPMG(x1,0,cols); 
    n.p100.y = n.p000.y;
    n.p100.z = n.p000.z; 
        
    n.p010.x = n.p000.x;
    n.p010.y = bcMapPeriodic_CPMG(y1,0,rows); 
    n.p010.z = n.p000.z; 
        
    n.p110.x = n.p100.x; 
    n.p110.y = n.p010.y; 
    n.p110.z = n.p000.z; 
        
    n.p001.x = n.p000.x; 
    n.p001.y = n.p000.y; 
    n.p001.z = bcMapPeriodic_CPMG(z1,0,depth);     
    
    n.p101.x = n.p100.x; 
    n.p101.y = n.p000.y; 
    n.p101.z = n.p001.z;    
    
    n.p011.x = n.p000.x; 
    n.p011.y = n.p010.y; 
    n.p011.z = n.p001.z;     
    
    n.p111.x = n.p100.x; 
    n.p111.y = n.p010.y; 
    n.p111.z = n.p001.z; 
}

__device__ void getNeighbors_mirror(Neighborhood &n, int x, int y, int z, int stride, uint shiftConverter, int cols, int rows, int depth)
{   
    int x0 = (x >> shiftConverter) - (int) ( ((x & (stride-1)) < (stride>>1)) ); // if x < x0, x0 = xc - 1
    int x1 = (x >> shiftConverter) + (int) (!((x & (stride-1)) < (stride>>1)) ); // if x > x0, x1 = xc + 1 
    int y0 = (y >> shiftConverter) - (int) ( ((y & (stride-1)) < (stride>>1)) ); // if y < y0, y0 = yc - 1
    int y1 = (y >> shiftConverter) + (int) (!((y & (stride-1)) < (stride>>1)) ); // if y > y0, y1 = yc + 1 
    int z0 = (z >> shiftConverter) - (int) ( ((z & (stride-1)) < (stride>>1)) ); // if z < z0, z0 = zc - 1
    int z1 = (z >> shiftConverter) + (int) (!((z & (stride-1)) < (stride>>1)) ); // if z > z0, z1 = zc + 1 
    
    n.p000.x = bcMapMirror_CPMG(x0,0,cols); 
    n.p000.y = bcMapMirror_CPMG(y0,0,rows); 
    n.p000.z = bcMapMirror_CPMG(z0,0,depth); 
    
    n.p100.x = bcMapMirror_CPMG(x1,0,cols); 
    n.p100.y = n.p000.y;
    n.p100.z = n.p000.z; 
        
    n.p010.x = n.p000.x;
    n.p010.y = bcMapMirror_CPMG(y1,0,rows); 
    n.p010.z = n.p000.z; 
        
    n.p110.x = n.p100.x; 
    n.p110.y = n.p010.y; 
    n.p110.z = n.p000.z; 
        
    n.p001.x = n.p000.x; 
    n.p001.y = n.p000.y; 
    n.p001.z = bcMapMirror_CPMG(z1,0,depth);     
    
    n.p101.x = n.p100.x; 
    n.p101.y = n.p000.y; 
    n.p101.z = n.p001.z;    
    
    n.p011.x = n.p000.x; 
    n.p011.y = n.p010.y; 
    n.p011.z = n.p001.z;     
    
    n.p111.x = n.p100.x; 
    n.p111.y = n.p010.y; 
    n.p111.z = n.p001.z; 
}

__device__ void getInterpCube(InterpCube &ic, Neighborhood &n, const double *field, int rowScale, int depthScale)
{    
    ic.i000 = getFieldIndex(n.p000.x,n.p000.y,n.p000.z,rowScale,depthScale);
    ic.c000 = field[ic.i000];

    ic.i100 = getFieldIndex(n.p100.x,n.p100.y,n.p100.z,rowScale,depthScale);
    ic.c100 = field[ic.i100];

    ic.i010 = getFieldIndex(n.p010.x,n.p010.y,n.p010.z,rowScale,depthScale);
    ic.c010 = field[ic.i010];

    ic.i110 = getFieldIndex(n.p110.x,n.p110.y,n.p110.z,rowScale,depthScale);
    ic.c110 = field[ic.i110];

    ic.i001 = getFieldIndex(n.p001.x,n.p001.y,n.p001.z,rowScale,depthScale);
    ic.c001 = field[ic.i001];

    ic.i101 = getFieldIndex(n.p101.x,n.p101.y,n.p101.z,rowScale,depthScale);
    ic.c101 = field[ic.i101];

    ic.i011 = getFieldIndex(n.p011.x,n.p011.y,n.p011.z,rowScale,depthScale);
    ic.c011 = field[ic.i011];

    ic.i111 = getFieldIndex(n.p111.x,n.p111.y,n.p111.z,rowScale,depthScale);
    ic.c111 = field[ic.i111];
}

__device__ double interpValues(InterpCube &ic, int x, int y, int z, int stride)
{
    // compensation for sc=0 cases:
    double comp = (double) (stride != 1);
    double xd = comp*(((double) ((x-(stride/2)) & (stride-1))) + 0.5)/stride; 
    double c00 = ic.c000*(1.0 - xd) + ic.c100*xd;
    double c01 = ic.c001*(1.0 - xd) + ic.c101*xd;
    double c10 = ic.c010*(1.0 - xd) + ic.c110*xd;
    double c11 = ic.c011*(1.0 - xd) + ic.c111*xd;
    
    double yd = comp*(((double) ((y-(stride/2)) & (stride-1))) + 0.5)/stride;
    double c0 = c00*(1.0-yd) + c10*yd;
    double c1 = c01*(1.0-yd) + c11*yd;
    
    double zd = comp*(((double) ((z-(stride/2)) & (stride-1))) + 0.5)/stride;
    double val = c0*(1.0-zd) + c1*zd;
    return val;
}

__device__ double triLinInterp_periodic(const double *field, int x, int y, int z, int stride, int shiftConverter, int cols, int rows, int depth, Neighborhood &n, InterpCube &ic)
{
    getNeighbors_periodic(n, x, y, z, stride, shiftConverter, cols, rows, depth);
    getInterpCube(ic, n, field, cols, cols*rows);
    return interpValues(ic, x, y, z, stride);    
}

__device__ double triLinInterp_mirror(const double *field, int x, int y, int z, int stride, int shiftConverter, int cols, int rows, int depth, Neighborhood &n, InterpCube &ic)
{
    getNeighbors_mirror(n, x, y, z, stride, shiftConverter, cols, rows, depth);
    getInterpCube(ic, n, field, cols, cols*rows);
    return interpValues(ic, x, y, z, stride);    
}

/*
    wrappers for kernel testing
*/

void NMR_cpmg::trilinearInterpolation()
{
    // CUDA event recorder to measure computation time in device
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // integer values for sizing issues
    uint numberOfWalkers = this->model.getNumberOfWalkers();
    uint bitBlockColumns = this->model.getBitBlock()->getBlockColumns();
    uint bitBlockRows = this->model.getBitBlock()->getBlockRows();
    uint numberOfBitBlocks = this->model.getBitBlock()->getNumberOfBlocks();
    int imageColumns = this->model.getBitBlock()->getImageColumns();
    int imageRows = this->model.getBitBlock()->getImageRows();
    int imageDepth = this->model.getBitBlock()->getImageDepth();
    uint shiftConverter = log2(this->model.getVoxelDivision());
    
    // THIS NEEDS TO BE REVISED LATER!!!
    double *field = (*this).getInternalFieldData();
    long fieldSize = (*this).getInternalFieldSize();
    
    // Copy bitBlock3D data from host to device (only once)
    // assign pointer to bitBlock datastructure
    uint64_t *bitBlock;
    bitBlock = this->model.getBitBlock()->getBlocks();
    uint64_t *d_bitBlock;
    
    // Host and Device memory data allocation
    // pointers used in host array conversion
    int *walker_px = MemAllocator::mallocIntArray(numberOfWalkers);
    int *walker_py = MemAllocator::mallocIntArray(numberOfWalkers);
    int *walker_pz = MemAllocator::mallocIntArray(numberOfWalkers);
    
    for (uint i = 0; i < numberOfWalkers; i++)
    {
        walker_px[i] = (*this->model.getWalkers())[i].getInitialPositionX();
        walker_py[i] = (*this->model.getWalkers())[i].getInitialPositionY();
        walker_pz[i] = (*this->model.getWalkers())[i].getInitialPositionZ();
    }

    // Declaration of pointers to device data arrays
    int *d_walker_px;
    int *d_walker_py;
    int *d_walker_pz;
    double *d_field;

    // Memory allocation in device for data arrays
    hipMalloc((void **)&d_bitBlock, numberOfBitBlocks * sizeof(uint64_t));
    hipMalloc((void **)&d_walker_px, numberOfWalkers * sizeof(int));
    hipMalloc((void **)&d_walker_py, numberOfWalkers * sizeof(int));
    hipMalloc((void **)&d_walker_pz, numberOfWalkers * sizeof(int));
    hipMalloc((void **)&d_field, fieldSize * sizeof(double));
    
    // Copy data from host to device
    hipMemcpy(d_bitBlock, bitBlock, numberOfBitBlocks * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_walker_px, walker_px, numberOfWalkers * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_walker_py, walker_py, numberOfWalkers * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_walker_pz, walker_pz, numberOfWalkers * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_field, field, fieldSize * sizeof(double), hipMemcpyHostToDevice);

    /*
        Kernel calls
    */
    uint threadsPerBlock = this->model.getRwnmrConfig().getThreadsPerBlock();
    uint blocksPerKernel = this->model.getRwnmrConfig().getBlocks();
    triLinInterp<<<1, 1>>>(d_walker_px, 
                           d_walker_py, 
                           d_walker_pz, 
                           d_field, 
                           d_bitBlock, 
                           bitBlockColumns, 
                           bitBlockRows, 
                           numberOfWalkers, 
                           imageColumns, 
                           imageRows, 
                           imageDepth, 
                           shiftConverter);

    hipDeviceSynchronize();

    /*
        End of routine
    */
    // free pointers in host
    free(walker_px);
    free(walker_py);
    free(walker_pz);
    
    // and direct them to NULL
    walker_px = NULL;
    walker_py = NULL;
    walker_pz = NULL;
    
    // also direct the bitBlock pointer created in this context
    // (original data is kept safe)
    bitBlock = NULL;
    field = NULL;

    // free device global memory
    hipFree(d_bitBlock);
    hipFree(d_walker_px);
    hipFree(d_walker_py);
    hipFree(d_walker_pz);
    hipFree(d_field);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Done.\nCpu/Gpu elapsed time: " << elapsedTime * 1.0e-3 << " s" << endl;
    hipDeviceReset();
}

void NMR_cpmg::bcMaps(int shiftConverter, int dimSize)
{
    /*
        Kernel calls
    */
    if(this->model.getRwnmrConfig().getBC() == "mirror")
        bcMap_test<true><<<1, 1>>>(shiftConverter, dimSize);
    else
        bcMap_test<false><<<1, 1>>>(shiftConverter, dimSize);
    
    hipDeviceSynchronize();   
    hipDeviceReset();
}

void NMR_cpmg::bcMapsExectime(int shiftConverter, int dimSize)
{    
    int bpk = 1024;
    int tpb = 1024;
    int size = tpb*bpk;
    bool printCondition = false;

    int *h_pos = MemAllocator::mallocIntArray(size);
    int *h_pos1 = MemAllocator::mallocIntArray(size);
    int *h_pos2 = MemAllocator::mallocIntArray(size);
    for(int i = 0; i < size; i++) h_pos[i] = i - (size/2);
    for(int i = 0; i < size; i++) h_pos1[i] = i - (size/2);
    for(int i = 0; i < size; i++) h_pos2[i] = i - (size/2);

    int *d_pos1;
    int *d_pos2;
    hipMalloc((void **)&d_pos1, size * sizeof(int));
    hipMemcpy(d_pos1, h_pos1, size * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_pos2, size * sizeof(int));
    hipMemcpy(d_pos2, h_pos2, size * sizeof(int), hipMemcpyHostToDevice);
    /*
        Kernel calls old implementation
    */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    if(this->model.getRwnmrConfig().getBC() == "mirror")
        bcMap_testExecTime<true,false><<<bpk,tpb>>>(d_pos1, size, shiftConverter, dimSize);
    else
        bcMap_testExecTime<false,false><<<bpk,tpb>>>(d_pos1, size, shiftConverter, dimSize);    
    hipDeviceSynchronize();
    hipMemcpy(h_pos1, d_pos1, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Old implementation: gpu elapsed time: " << elapsedTime * 1.0e-3 << " s" << endl;   
    
    /*
        Kernel calls new implementation
    */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    if(this->model.getRwnmrConfig().getBC() == "mirror")
        bcMap_testExecTime<true,true><<<bpk,tpb>>>(d_pos2, size, shiftConverter, dimSize);
    else
        bcMap_testExecTime<false,true><<<bpk,tpb>>>(d_pos2, size, shiftConverter, dimSize);
    hipDeviceSynchronize();
    hipMemcpy(h_pos2, d_pos2, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);    
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "New implementation: gpu elapsed time: " << elapsedTime * 1.0e-3 << " s" << endl;   
    
    for(int i = 0; i < size; i++)
        if(printCondition) cout << "pos: " << h_pos[i] << "\told: " << h_pos1[i] << "\tnew: " << h_pos2[i] << endl;

    free(h_pos); h_pos=NULL;
    free(h_pos1); h_pos1=NULL;
    free(h_pos2); h_pos2=NULL;
    hipFree(d_pos1);
    hipFree(d_pos2);
    hipDeviceReset();
}
