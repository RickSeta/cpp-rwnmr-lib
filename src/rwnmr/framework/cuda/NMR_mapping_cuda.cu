#include "hip/hip_runtime.h"
#include "NMR_mapping_cuda.h"

// function to call GPU kernel to execute
// walker's "map" method in Graphics Processing Unit
void Model::mapSimulation_CUDA_3D_histograms(bool reset)
{   
    string bc = (*this).getBoundaryCondition();

    cout << "- starting 3DRW-Mapping simulation (in GPU) [bc:" << bc << "]...";
        
    // reset walkers
    if(reset)
    {
        if(this->rwNMR_config.getOpenMPUsage())
        {
            // set omp variables for parallel loop throughout walker list
            const int num_cpu_threads = omp_get_max_threads();
            const int loop_size = this->getWalkers()->size();
            int loop_start, loop_finish;

            #pragma omp parallel shared(walkers) private(loop_start, loop_finish) 
            {
                const int thread_id = omp_get_thread_num();
                ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                loop_start = looper.getStart();
                loop_finish = looper.getFinish(); 

                for (uint id = loop_start; id < loop_finish; id++)
                {
                    (*this->getWalkers())[id].resetPosition();
                    (*this->getWalkers())[id].resetSeed();
                    (*this->getWalkers())[id].resetCollisions();
                    (*this->getWalkers())[id].resetTCollisions();
                }
            }
        } else
        {
            for (uint id = 0; id < this->getWalkers()->size(); id++)
            {
                (*this->getWalkers())[id].resetPosition();
                (*this->getWalkers())[id].resetSeed();
                (*this->getWalkers())[id].resetCollisions();
                (*this->getWalkers())[id].resetTCollisions();
            }
        }
    }

    // CUDA event recorder to measure computation time in device
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // integer values
    uint bitBlockColumns = this->bitBlock->getBlockColumns();
    uint bitBlockRows = this->bitBlock->getBlockRows();
    uint numberOfBitBlocks = this->bitBlock->getNumberOfBlocks();
    uint numberOfWalkers = this->numberOfWalkers;
    int map_columns = this->bitBlock->getImageColumns();
    int map_rows = this->bitBlock->getImageRows();
    int map_depth = this->bitBlock->getImageDepth();
    uint shiftConverter = log2(this->voxelDivision);

    // define parameters for CUDA kernel launch: blockDim, gridDim etc
    uint threadsPerBlock = this->rwNMR_config.getThreadsPerBlock();
    uint blocksPerKernel = this->rwNMR_config.getBlocks();
    uint walkersPerKernel = threadsPerBlock * blocksPerKernel;
    if (numberOfWalkers < walkersPerKernel)
    {
        blocksPerKernel = (int)ceil((double)(numberOfWalkers) / (double)(threadsPerBlock));
        walkersPerKernel = threadsPerBlock * blocksPerKernel;
    }
    uint numberOfWalkerPacks = (numberOfWalkers / walkersPerKernel) + 1;
    uint lastWalkerPackSize = numberOfWalkers % walkersPerKernel;

    // bitBlock3D host to device copy
    // assign pointer to bitBlock datastructure
    uint64_t *bitBlock;
    bitBlock = this->bitBlock->getBlocks();

    // copy host bitblock data to temporary host arrays
    uint64_t *d_bitBlock;
    hipMalloc((void **)&d_bitBlock, numberOfBitBlocks * sizeof(uint64_t));
    hipMemcpy(d_bitBlock, bitBlock, numberOfBitBlocks * sizeof(uint64_t), hipMemcpyHostToDevice);

    // Device memory allocation
    // pointers used in array conversion and their host memory allocation
    int *walker_px = MemAllocator::mallocIntArray(walkersPerKernel);
    int *walker_py = MemAllocator::mallocIntArray(walkersPerKernel);
    int *walker_pz = MemAllocator::mallocIntArray(walkersPerKernel);
    uint *collisions = MemAllocator::mallocUIntArray(walkersPerKernel);
    uint64_t *seed = MemAllocator::mallocUInt64Array(walkersPerKernel);

    // Device memory allocation
    // Declaration of device data arrays
    int *d_walker_px;
    int *d_walker_py;
    int *d_walker_pz;
    uint *d_collisions;
    uint64_t *d_seed;

    // alloc memory in device for data arrays
    hipMalloc((void **)&d_walker_px, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_walker_py, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_walker_pz, walkersPerKernel * sizeof(int));
    hipMalloc((void **)&d_collisions, walkersPerKernel * sizeof(uint));
    hipMalloc((void **)&d_seed, walkersPerKernel * sizeof(uint64_t));

    // initialize histograms
    (*this).initHistogramList();

    // loop throughout histogram list
    for(int hst_ID = 0; hst_ID < this->histogramList.size(); hst_ID++)
    {
        // set steps for each histogram
        uint eBegin = this->histogramList[hst_ID].getFirstEcho();
        uint eEnd = this->histogramList[hst_ID].getLastEcho();
        uint steps = this->stepsPerEcho * (eEnd - eBegin);

        // create a steps bucket
        uint stepsLimit = this->rwNMR_config.getMaxRWSteps();
        uint stepsSize = steps/stepsLimit;
        vector<uint> stepsList;
        for(uint idx = 0; idx < stepsSize; idx++)
        {
            stepsList.push_back(stepsLimit);
        }
        // charge rebalance
        if((steps % stepsLimit) > 0)
        {
            stepsSize++;
            stepsList.push_back(steps%stepsLimit);
        } 

        for (uint packId = 0; packId < (numberOfWalkerPacks - 1); packId++)
        {
            // set offset in walkers vector
            uint packOffset = packId * walkersPerKernel;
    
            // Host data copy
            // copy original walkers' data to temporary host arrays
            if(this->rwNMR_config.getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = walkersPerKernel;
                int loop_start, loop_finish;

                #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, collisions, seed, walkers) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        walker_px[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionX();
                        walker_py[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionY();
                        walker_pz[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionZ();
                        collisions[id] = 0;
                        seed[id] = (*this->getWalkers())[id + packOffset].getCurrentSeed();
                    }
                }
            } else
            {
                for (uint id = 0; id < walkersPerKernel; id++)
                {
                    walker_px[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionX();
                    walker_py[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionY();
                    walker_pz[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionZ();
                    collisions[id] = 0;
                    seed[id] = (*this->getWalkers())[id + packOffset].getCurrentSeed();
                }
            }
    
            // Device data copy
            // copy host data to device
            hipMemcpy(d_walker_px, walker_px, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_walker_py, walker_py, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_walker_pz, walker_pz, walkersPerKernel * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_collisions, collisions, walkersPerKernel * sizeof(uint), hipMemcpyHostToDevice);
            hipMemcpy(d_seed, seed, walkersPerKernel * sizeof(uint64_t), hipMemcpyHostToDevice);
    
            //////////////////////////////////////////////////////////////////////
            // Launch kernel for GPU computation
            // kernel "map" launch
            for(uint sIdx = 0; sIdx < stepsList.size(); sIdx++)
            {
                if(bc == "mirror")
                {
                    rwMap<true><<<blocksPerKernel, threadsPerBlock>>>(d_walker_px,
                                                                      d_walker_py,
                                                                      d_walker_pz,
                                                                      d_collisions,
                                                                      d_seed,
                                                                      d_bitBlock,
                                                                      bitBlockColumns,
                                                                      bitBlockRows,
                                                                      walkersPerKernel,
                                                                      stepsList[sIdx],
                                                                      map_columns,
                                                                      map_rows,
                                                                      map_depth,
                                                                      shiftConverter);
                }
                else 
                {
                    rwMap<false><<<blocksPerKernel, threadsPerBlock>>>(d_walker_px,
                                                                       d_walker_py,
                                                                       d_walker_pz,
                                                                       d_collisions,
                                                                       d_seed,
                                                                       d_bitBlock,
                                                                       bitBlockColumns,
                                                                       bitBlockRows,
                                                                       walkersPerKernel,
                                                                       stepsList[sIdx],
                                                                       map_columns,
                                                                       map_rows,
                                                                       map_depth,
                                                                       shiftConverter);
                
                } 
                
                hipDeviceSynchronize();
            }

            // Host data copy
            // copy device data to host
            hipMemcpy(collisions, d_collisions, walkersPerKernel * sizeof(uint), hipMemcpyDeviceToHost);            
            hipMemcpy(walker_px, d_walker_px, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_py, d_walker_py, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_pz, d_walker_pz, walkersPerKernel * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(seed, d_seed, walkersPerKernel * sizeof(uint64_t), hipMemcpyDeviceToHost);
            
            // copy collisions host data to class members
            if(this->rwNMR_config.getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = walkersPerKernel;
                int loop_start, loop_finish;

                #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, collisions, seed, walkers) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        (*this->getWalkers())[id + packOffset].setCollisions(collisions[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]); 
                        (*this->getWalkers())[id + packOffset].setCurrentSeed(seed[id]);
                    }
                }
            } else
            {
                
                for (uint id = 0; id < walkersPerKernel; id++)
                {
                    (*this->getWalkers())[id + packOffset].setCollisions(collisions[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]); 
                    (*this->getWalkers())[id + packOffset].setCurrentSeed(seed[id]);
                }
            }
    
            
        }
    
        if (lastWalkerPackSize > 0)
        { 
            // last pack is done explicitly
            // set offset in walkers vector
            uint packOffset = (numberOfWalkerPacks - 1) * walkersPerKernel;
    
            // Host data copy
            // copy original walkers' data to temporary host arrays
            if(this->rwNMR_config.getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = lastWalkerPackSize;
                int loop_start, loop_finish;

                #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, collisions, seed, walkers) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        walker_px[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionX();
                        walker_py[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionY();
                        walker_pz[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionZ();
                        collisions[id] = 0;
                        seed[id] = (*this->getWalkers())[id + packOffset].getCurrentSeed();
                    }
                }
            } else
            {
                for (uint id = 0; id < lastWalkerPackSize; id++)
                {
                    walker_px[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionX();
                    walker_py[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionY();
                    walker_pz[id] = (*this->getWalkers())[id + packOffset].getCurrentPositionZ();
                    collisions[id] = 0;
                    seed[id] = (*this->getWalkers())[id + packOffset].getCurrentSeed();
                }
            }
    
            // Device data copy
            // copy host data to device
            hipMemcpy(d_walker_px, walker_px, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_walker_py, walker_py, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_walker_pz, walker_pz, lastWalkerPackSize * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_collisions, collisions, lastWalkerPackSize * sizeof(uint), hipMemcpyHostToDevice);
            hipMemcpy(d_seed, seed, lastWalkerPackSize * sizeof(uint64_t), hipMemcpyHostToDevice);
    
            //////////////////////////////////////////////////////////////////////
            // Launch kernel for GPU computation
            // kernel "map" launch
            for(uint sIdx = 0; sIdx < stepsList.size(); sIdx++)
            {
                if(bc == "mirror")
                {
                    rwMap<true><<<blocksPerKernel, threadsPerBlock>>>(d_walker_px,
                                                                      d_walker_py,
                                                                      d_walker_pz,
                                                                      d_collisions,
                                                                      d_seed,
                                                                      d_bitBlock,
                                                                      bitBlockColumns,
                                                                      bitBlockRows,
                                                                      lastWalkerPackSize,
                                                                      stepsList[sIdx],
                                                                      map_columns,
                                                                      map_rows,
                                                                      map_depth,
                                                                      shiftConverter);
                }
                else 
                {
                    rwMap<false><<<blocksPerKernel, threadsPerBlock>>>(d_walker_px,
                                                                       d_walker_py,
                                                                       d_walker_pz, 
                                                                       d_collisions, 
                                                                       d_seed, 
                                                                       d_bitBlock, 
                                                                       bitBlockColumns, 
                                                                       bitBlockRows, 
                                                                       lastWalkerPackSize, 
                                                                       stepsList[sIdx], 
                                                                       map_columns, 
                                                                       map_rows, 
                                                                       map_depth, 
                                                                       shiftConverter);
                }
                hipDeviceSynchronize();
            }
    
            // Host data copy
            // copy device data to host
            hipMemcpy(collisions, d_collisions, lastWalkerPackSize * sizeof(uint), hipMemcpyDeviceToHost);
            hipMemcpy(walker_px, d_walker_px, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_py, d_walker_py, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(walker_pz, d_walker_pz, lastWalkerPackSize * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(seed, d_seed, lastWalkerPackSize * sizeof(uint64_t), hipMemcpyDeviceToHost);
            
    
            // copy collisions host data to class members
            if(this->rwNMR_config.getOpenMPUsage())
            {
                // set omp variables for parallel loop throughout walker list
                const int num_cpu_threads = omp_get_max_threads();
                const int loop_size = lastWalkerPackSize;
                int loop_start, loop_finish;

                #pragma omp parallel shared(packOffset, walker_px, walker_py, walker_pz, collisions, seed, walkers) private(loop_start, loop_finish) 
                {
                    const int thread_id = omp_get_thread_num();
                    ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                    loop_start = looper.getStart();
                    loop_finish = looper.getFinish(); 

                    for (uint id = loop_start; id < loop_finish; id++)
                    {
                        (*this->getWalkers())[id + packOffset].setCollisions(collisions[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                        (*this->getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]); 
                        (*this->getWalkers())[id + packOffset].setCurrentSeed(seed[id]);
                    }
                }
            } else
            {
                
                for (uint id = 0; id < lastWalkerPackSize; id++)
                {
                    (*this->getWalkers())[id + packOffset].setCollisions(collisions[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionX(walker_px[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionY(walker_py[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentPositionZ(walker_pz[id]);
                    (*this->getWalkers())[id + packOffset].setCurrentSeed(seed[id]);
                }
            }
        }

        // create histogram
        (*this).createHistogram(hst_ID, steps);

        // reset collision count, but keep summation in alternative count
        if(this->rwNMR_config.getOpenMPUsage())
        {
            // set omp variables for parallel loop throughout walker list
            const int num_cpu_threads = omp_get_max_threads();
            const int loop_size = this->numberOfWalkers;
            int loop_start, loop_finish;

            #pragma omp parallel shared(walkers) private(loop_start, loop_finish) 
            {
                const int thread_id = omp_get_thread_num();
                ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
                loop_start = looper.getStart();
                loop_finish = looper.getFinish(); 

                for (uint id = loop_start; id < loop_finish; id++)
                {
                    (*this->getWalkers())[id].setTCollisions((*this->getWalkers())[id].getTCollisions() + (*this->getWalkers())[id].getCollisions());
                    (*this->getWalkers())[id].resetCollisions();
                }
            }
        } else
        {
            for (uint id = 0; id < this->numberOfWalkers; id++)
            {
                (*this->getWalkers())[id].setTCollisions((*this->getWalkers())[id].getTCollisions() + (*this->getWalkers())[id].getCollisions());
                (*this->getWalkers())[id].resetCollisions();
            }
        }
    }
    // histogram loop is finished

    // recover walkers collisions from total sum and create a global histogram
    if(this->rwNMR_config.getOpenMPUsage())
    {
        // set omp variables for parallel loop throughout walker list
        const int num_cpu_threads = omp_get_max_threads();
        const int loop_size = this->numberOfWalkers;
        int loop_start, loop_finish;

        #pragma omp parallel shared(walkers) private(loop_start, loop_finish) 
        {
            const int thread_id = omp_get_thread_num();
            ThreadsBalancer looper(thread_id, num_cpu_threads, loop_size);
            loop_start = looper.getStart();
            loop_finish = looper.getFinish(); 

            for (uint id = loop_start; id < loop_finish; id++)
            {
                (*this->getWalkers())[id].setCollisions((*this->getWalkers())[id].getTCollisions());
            }
        }

    } else
    {
        for (uint id = 0; id < this->numberOfWalkers; id++)
        {
            (*this->getWalkers())[id].setCollisions((*this->getWalkers())[id].getTCollisions());   
        }
    }

    // create collision histogram
    (*this).createHistogram();

    // free pointers in host
    free(walker_px);
    free(walker_py);
    free(walker_pz);
    free(collisions);
    free(seed);

    // and direct them to NULL
    walker_px = NULL;
    walker_py = NULL;
    walker_pz = NULL;
    collisions = NULL;
    seed = NULL;

    // also direct the bitBlock pointer created in this context
    // (original data is kept safe)
    bitBlock = NULL;

    // free device global memory
    hipFree(d_walker_px);
    hipFree(d_walker_py);
    hipFree(d_walker_pz);
    hipFree(d_collisions);
    hipFree(d_seed);
    hipFree(d_bitBlock);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Done.\nelapsed time: " << elapsedTime * 1.0e-3 << " seconds" << endl;    
}

/*

Device functions

*/

__device__ direction computeNextDirection_MAP(uint64_t &seed)
{
    // generate random number using xorshift algorithm
    xorshift64_state xor_state;
    xor_state.a = seed;
    seed = xorShift64_MAP(&xor_state);
    uint64_t rand = seed;

    // set direction based on the random number
    direction nextDirection = (direction)(mod6_MAP(rand) + 1);
    return nextDirection;
}

__device__ uint64_t xorShift64_MAP(struct xorshift64_state *state)
{
    uint64_t x = state->a;
    x ^= x << 13;
    x ^= x >> 7;
    x ^= x << 17;
    return state->a = x;
}

__device__ uint64_t mod6_MAP(uint64_t a)
{
    while (a > 11)
    {
        int s = 0; /* accumulator for the sum of the digits */
        while (a != 0)
        {
            s = s + (a & 7);
            a = (a >> 2) & -2;
        }
        a = s;
    }
    /* note, at this point: a < 12 */
    if (a > 5)
        a = a - 6;
    return a;
}

__device__ void computeNextPosition_MAP(int &walker_px,
                                       int &walker_py,
                                       int &walker_pz,
                                       direction nextDirection,
                                       int &next_x,
                                       int &next_y,
                                       int &next_z)
{
    next_x = walker_px;
    next_y = walker_py;
    next_z = walker_pz;

    switch (nextDirection)
    {
    case North:
        next_y = next_y - 1;
        break;

    case West:
        next_x = next_x - 1;
        break;

    case South:
        next_y = next_y + 1;
        break;

    case East:
        next_x = next_x + 1;
        break;

    case Up:
        next_z = next_z + 1;
        break;

    case Down:
        next_z = next_z - 1;
        break;
    }
}

__device__ bool checkNextPosition_MAP(int next_x,
                                     int next_y,
                                     int next_z,
                                     const uint64_t *bitBlock,
                                     const int bitBlockColumns,
                                     const int bitBlockRows)
{
    int blockIndex = findBlockIndex_MAP(next_x, next_y, next_z, bitBlockColumns, bitBlockRows);
    int nextBit = findBitIndex_MAP(next_x, next_y, next_z);
    uint64_t nextBlock = bitBlock[blockIndex];

    return (!checkIfBlockBitIsWall_MAP(nextBlock, nextBit));
};

__device__ int findBlockIndex_MAP(int next_x, int next_y, int next_z, int bitBlockColumns, int bitBlockRows)
{
    // "x >> 2" is like "x / 4" in bitwise operation
    int block_x = next_x >> 2;
    int block_y = next_y >> 2;
    int block_z = next_z >> 2;
    int blockIndex = block_x + block_y * bitBlockColumns + block_z * (bitBlockColumns * bitBlockRows);

    return blockIndex;
}

__device__ int findBitIndex_MAP(int next_x, int next_y, int next_z)
{
    // "x & (n - 1)" is lise "x % n" in bitwise operation
    int bit_x = next_x & (COLUMNSPERBLOCK3D - 1);
    int bit_y = next_y & (ROWSPERBLOCK3D - 1);
    int bit_z = next_z & (DEPTHPERBLOCK3D - 1);
    // "x << 3" is like "x * 8" in bitwise operation
    int bitIndex = bit_x + (bit_y << 2) + ((bit_z << 2) << 2);

    return bitIndex;
}

__device__ bool checkIfBlockBitIsWall_MAP(uint64_t nextBlock, int nextBit)
{
    return ((nextBlock >> nextBit) & 1ull);
}

__device__ int convertLocalToGlobal_MAP(int _localPos, uint _shiftConverter)
{
    return (_localPos >> _shiftConverter);
}

__device__ int bcMapPeriodic_MAP(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos;
    globalPos = ( convertLocalToGlobal_MAP(_localPos, _shiftConverter) ) % _dimSize;
    globalPos += (((int) (globalPos < 0)) * _dimSize);
    return globalPos;
}

__device__ int bcMapMirror_MAP(int _localPos, uint _shiftConverter, int _dimSize)
{
    int globalPos = convertLocalToGlobal_MAP(_localPos, _shiftConverter);
    int imgPos = (globalPos % _dimSize); 
    imgPos += (((int) (imgPos < 0)) * _dimSize);
    int mirror = ((isPositive_MAP(globalPos)*globalPos) + (((int) !(globalPos > 0)) * (-globalPos-1+_dimSize)))/_dimSize; 
    mirror = (mirror & 1);
    return (((mirror + 1) & 1) * imgPos) + (mirror * (_dimSize - 1 - imgPos));
}

__device__ int isPositive_MAP(int x) 
{
   return !((x&(1<<31)) | !x);
}

__device__ int isZero_MAP(int x) 
{
   return !x;
}
